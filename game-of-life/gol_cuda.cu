#include "hip/hip_runtime.h"
#include "gol_parallel.h"
#include <hip/hip_runtime.h>

__global__ void gol_kernal(bool *cells_d, bool *cells_new_d, int n, int m, int block_width, int block_height, int thread_width, int thread_height){
  int i, j;
  int startRow = blockIdx.y * block_height + threadIdx.y * thread_height;
  int startCol = blockIdx.x * block_width + threadIdx.x * thread_width;
  int endRow = min(startRow + thread_height, n); // End should probably be min of start + height and n
  int endCol = min(startCol + thread_width, m);
  int neighbors;

  for(i = startRow; i < endRow; i++){
    for(j = startCol; j < endCol; j++){
      neighbors = 0;

      if(i>0 && j>0) neighbors += cells_d[(i-1) * m + j-1] ? 1 : 0;
      if(i>0) neighbors += cells_d[(i-1) * m + j] ? 1 : 0;
      if(i>0 && j<m-1) neighbors += cells_d[(i-1) * m + j+1] ? 1 : 0;
      if(j>0) neighbors += cells_d[i * m + j-1] ? 1 : 0;
      if(j<m-1) neighbors += cells_d[i * m + j+1] ? 1 : 0;
      if(i<n-1 && j>0) neighbors += cells_d[(i+1) * m + j-1] ? 1 : 0;
      if(i<n-1) neighbors += cells_d[(i+1) * m + j] ? 1 : 0;
      if(i<n-1 && j<m-1) neighbors += cells_d[(i+1) * m + j+1] ? 1 : 0;

      cells_new_d[i * m + j] = neighbors == 3 || (cells_d[i * m + j] && neighbors == 2);
    }
  }
}

void gol_cuda(char* cells, int n, int m, int iterations, int xBlockDim, int yBlockDim, int xThreadDim, int yThreadDim){
  int i;
  int size = n * m * sizeof(bool);
  bool *cells_d, *cells_new_d, *tmp;

  hipMalloc(&cells_d, size);
  hipMemcpy(cells_d, cells, size, hipMemcpyHostToDevice);

  hipMalloc(&cells_new_d, size);
  hipMemcpy(cells_new_d, cells, size, hipMemcpyHostToDevice);

  // Kernal implementation
  dim3 dimGrid(xThreadDim, yThreadDim);
  dim3 dimBlock(xBlockDim, yBlockDim);

  int block_width = m % xBlockDim ? m / xBlockDim + 1 : m / xBlockDim;
  int block_height = n % yBlockDim ? n / yBlockDim + 1 : n / yBlockDim;
  int thread_width = block_width % xThreadDim ? block_width / xThreadDim + 1 : block_width / xThreadDim;
  int thread_height = block_height % yThreadDim ? block_height / yThreadDim + 1 : block_height / yThreadDim;

  for(i = 0; i < iterations; i++){
    gol_kernal<<<dimBlock, dimGrid>>>(cells_d, cells_new_d, n, m, block_width, block_height, thread_width, thread_height);

    tmp = cells_d;
    cells_d = cells_new_d;
    cells_new_d = tmp;
  }


  hipMemcpy(cells, cells_d, size, hipMemcpyDeviceToHost);
  hipFree(cells_d); hipFree(cells_new_d);

}